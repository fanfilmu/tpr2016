#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include "helper_functions.h"

#ifndef max
#define max(a,b) (((a) (b)) ? (a) : (b))
#define min(a,b) (((a) < (b)) ? (a) : (b))
#endif

typedef unsigned long long u64;

__global__ void add (int *a, int *b, int *c, u64 N, u64 offset) {
  u64 tid = blockIdx.x * blockDim.x + threadIdx.x + offset;
  if(tid < N) {
    c[tid] = a[tid] + b[tid];
  }
}

void cpu_add (int *a, int *b, int *c, u64 N) {
  u64 i;
  for (i = 0; i < N; i++) {
    c[i] = a[i] + b[i];
  }
}

int main(int argc, char** argv) {
  if (argc < 2) {
    printf("Usage: ./vectors N [block_size] [grid_size]\n");
    exit(-1);
  }

  u64 N = atoll(argv[1]);

  int* host_a = (int*)malloc(N * sizeof(int));
  int* host_b = (int*)malloc(N * sizeof(int));
  int* host_c = (int*)malloc(N * sizeof(int));
  int* host_r = (int*)malloc(N * sizeof(int));

  int *dev_a, *dev_b, *dev_c;

  hipMalloc((void**)&dev_a, N * sizeof(int));
  hipMalloc((void**)&dev_b, N * sizeof(int));
  hipMalloc((void**)&dev_c, N * sizeof(int));

  for (int i = 0; i < N; i++) {
    host_a[i] = i;
    host_b[i] = i * 2;
  }

  StopWatchInterface *timer=NULL;

  hipMemcpy(dev_a, host_a, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, host_b, N * sizeof(int), hipMemcpyHostToDevice);

  u64 block_size, grid_size;

  if (argc >= 3) {
    block_size = atoll(argv[2]);
  } else {
    block_size = 1024;
  }

  if (argc >= 4) {
    grid_size = atoll(argv[3]);
  } else {
    grid_size = min((int)ceil((double)N / block_size), 65535);
  }

  u64 offset = 0;

  sdkCreateTimer(&timer);
  sdkResetTimer(&timer);
  sdkStartTimer(&timer);

  while (offset < N) {
    add <<<grid_size,block_size>>> (dev_a, dev_b, dev_c, N, offset);
    offset += block_size * grid_size;
  }

  hipDeviceSynchronize();
  sdkStopTimer(&timer);
  float time = sdkGetTimerValue(&timer);

  hipMemcpy(host_c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
  sdkDeleteTimer(&timer);

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  struct timeval tval_before, tval_after, tval_result;
  gettimeofday(&tval_before, NULL);

  cpu_add(host_a, host_b, host_r, N);

  gettimeofday(&tval_after, NULL);
  timersub(&tval_after, &tval_before, &tval_result);
  double cpu_time = 1000.0 * (long int)tval_result.tv_sec + (long int)tval_result.tv_usec / 1000.0;
  u64 n;
  u64 errors = 0;  

  for (n = 0; n < N; n++) {
    if (host_c[n] != host_r[n]) {
      errors += 1;
    }
  }

  printf("%ld,%d,%d,%lf,%lf,%ld\n", N, block_size, grid_size, cpu_time, time, errors);

  return 0;
}
